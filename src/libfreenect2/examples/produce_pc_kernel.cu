#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>
 #include "hip/device_functions.h"
 #include "produce_pc_kernel.h"
 #include <stdio.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include ""
#include <hip/hip_runtime_api.h>
#include <string.h> // memcpy
#include <cstdlib>
#include <signal.h>
#include <time.h>

texture<int, 1, hipReadModeElementType> texref0, texref2;
texture<float, 1, hipReadModeElementType> texref1;
const size_t imageSize1 = 4 * 424 * 512;
__global__ void kernel2(const float* _depth, const float* _rgba, const int _width, const int _height, const float* cc, float *_cloud)//(const float* _depth, const float* _rgb, const int _width, const int _height, const int table_x[], const int table_y[], float *_cloud)
{

    int c = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int r = threadIdx.y + blockIdx.y * blockDim.y;//0-423

    const int index = (c) + (r) * _width;//0--512*424-1=217087
    const float cx = cc[0];
    const float cy = cc[1];
    const float fx = cc[2];
    const float fy = cc[3];
    if ( c < _width )
    {
        if ( r < _height)
        {
            const float &depth_v = _depth[index]; //scaling factor, so that value of 1 is one meter.
            _cloud[index*8+0] = (c-cx) * (fx) * depth_v;
            _cloud[index*8+1] = (r-cy) * (fy) * depth_v;
            _cloud[index*8+2] = depth_v;
            _cloud[index*8+4] = _rgba[index];
        }
     }
    __syncthreads();
  }

__global__ void kernel3(const float* d_dis_depth, const unsigned int* d_raw_rgb, unsigned int* d_register, const float* cc, float *_cloud)//, const float* cc, float *_cloud
{
    int c = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int r = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int index = (c) + (r) * 512;//0--512*424-1=217087
    int depth_to_c_off[512*424];
//    int offset = threadIdx.x + threadIdx.y * 32;
    const float &cx = cc[0];
    const float &cy = cc[1];
    const float &fx = cc[2];
    const float &fy = cc[3];
    if ( r < 424 )
    {
        if ( c < 512)
        {
            const int &id = tex1Dfetch(texref0,index);
            const float z = id <0 ? 0.02f : d_dis_depth[id];
            const int c_off = (z>0)*((tex1Dfetch(texref1,index) + __fdividef(52.0f,z)) * 1081.372070f + 960.0f + tex1Dfetch(texref2,index) * 1920);
            depth_to_c_off[index] = ((z>0)&&(id>=0)&&(c_off>=0)&&(c_off<1920 * 1080)) * (c_off+1) + (-1);
            const float &depth_v = d_dis_depth[id] * (id>=0);
            _cloud[index*8+0] = (c-cx) * (fx) * depth_v;
            _cloud[index*8+1] = (r-cy) * (fy) * depth_v;
            _cloud[index*8+2] = depth_v;
            const int &rgb_val = depth_to_c_off[index] < 0 ? 0 : d_raw_rgb[depth_to_c_off[index]];

            d_register[index] = rgb_val;

            u_char* rgba = (u_char*) ( _cloud + index*8+4 );
            const u_char* bgra = (const u_char*) &rgb_val;
            rgba[0] = bgra[2];
            rgba[1] = bgra[1];
            rgba[2] = bgra[0];
        }
    }
    __syncthreads();
}

void registration_kernel(const float* d_dis_depth, const unsigned int* d_raw_rgb, unsigned int* d_register, const int* d_map_dist, const float* d_map_x, const int* d_map_yi, const float* cc, float *_cloud)//, const float* cc, float *_cloud
{
    hipBindTexture(0,texref0,d_map_dist,imageSize1);
    hipBindTexture(0,texref1,d_map_x,imageSize1);
    hipBindTexture(0,texref2,d_map_yi,imageSize1);
    dim3 threadsPerBlock(32, 4);
    dim3 Grid((512+threadsPerBlock.x-1)/threadsPerBlock.x, (424+threadsPerBlock.y-1)/threadsPerBlock.y);
    kernel3<<< Grid, threadsPerBlock >>>(d_dis_depth, d_raw_rgb, d_register, cc, _cloud);
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    hipUnbindTexture(texref0);
    hipUnbindTexture(texref1);
    hipUnbindTexture(texref2);

}
void produce_pc_kernel2(const float* _depth, const float* _rgba, const int _width, const int _height, const float* c, float *_cloud)//(const float* _depth, const float* _rgb, const int _width, const int _height, const int table_x[], const int table_y[], float *_cloud)
{
    dim3 threadsPerBlock(32, 8);
    dim3 Grid((_width+threadsPerBlock.x-1)/threadsPerBlock.x, (_height+threadsPerBlock.y-1)/threadsPerBlock.y);
    kernel2<<< Grid, threadsPerBlock >>>(_depth, _rgba, _width, _height, c, _cloud);
}
