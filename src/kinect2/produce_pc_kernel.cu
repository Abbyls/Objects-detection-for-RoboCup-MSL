#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>
 #include <hip/hip_runtime.h>
 #include "hip/device_functions.h"
 #include "produce_pc_kernel.h"
 #include <stdio.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include ""
#include <hip/hip_runtime_api.h>
#include <string.h> // memcpy
#include <cstdlib>
#include <signal.h>
#include <time.h>
#include <math.h>
#include <thrust/device_ptr.h>
#include <algorithm>
#include <vector>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
//#include "Protonect.h"
//#include <af/hip/hip_runtime.h>
//#include <arrayfire.h>
#include <opencv2/opencv.hpp>
#include <LaterMethods.h>
#define PI 3.1415

using namespace std;
texture<int, 1, hipReadModeElementType> texref0, texref2;
texture<float, 1, hipReadModeElementType> texref1;
const size_t imageSize1 = 4 * 424 * 512;
dim3 threadsPerBlock(32, 32);
dim3 threadsPerBlock1(1, 400);
dim3 Grid1((400+threadsPerBlock1.x-1)/threadsPerBlock1.x, (400+threadsPerBlock1.y-1)/threadsPerBlock1.y);
dim3 Grid_((400+threadsPerBlock.x-1)/threadsPerBlock.x, (240+threadsPerBlock.y-1)/threadsPerBlock.y);
dim3 Grid((512+threadsPerBlock.x-1)/threadsPerBlock.x, (424+threadsPerBlock.y-1)/threadsPerBlock.y);

struct is_zero
{
    __host__ __device__ bool operator()(const int x)
    {
        return (x==0);
    }
};

__global__ void peng_zhang(unsigned char * pj,unsigned char *temp_pj,size_t width, size_t height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int j = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int idex = (i) + (j) * width;//0--512*424-1=217087
    int neighbors[8] = {(i-1) + (j-1) * width, (i) + (j-1) * width,(i+1) + (j-1) * width,(i-1) + (j) * width, (i+1) + (j) * width,(i-1) + (j+1) * width,(i) + (j+1) * width,(i+1) + (j+1) * width};
    if ( j < height )
    {
         if ( i < width)
        {
            if (temp_pj[idex]==0)//目标像素
            {
                if (j==0)//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[4]]>0)||(temp_pj[neighbors[6]]>0)||(temp_pj[neighbors[7]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;

                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[3]]>0)||(temp_pj[neighbors[5]]>0)||(temp_pj[neighbors[6]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=3;k<8;k++)
                        {
                            if (temp_pj[neighbors[k]]>0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=1;
                                    break;
                                }
                            }
                        }
                    }
                }
                if (j==(height-1))//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[1]]>0)||(temp_pj[neighbors[2]]>0)||(temp_pj[neighbors[4]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[0]]>0)||(temp_pj[neighbors[1]]>0)||(temp_pj[neighbors[3]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=0;k<5;k++)
                        {
                            if (temp_pj[neighbors[k]]>0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=1;
                                    break;
                                }
                            }
                        }
                    }
                }

                if ((j>0)&&(j<(height-1)))//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[1]]>0)||(temp_pj[neighbors[2]]>0)||(temp_pj[neighbors[4]]>0)||(temp_pj[neighbors[6]]>0)||(temp_pj[neighbors[7]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[0]]>0)||(temp_pj[neighbors[1]]>0)||(temp_pj[neighbors[3]]>0)||(temp_pj[neighbors[5]]>0)||(temp_pj[neighbors[6]]>0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=0;k<8;k++)
                        {
                            if (temp_pj[neighbors[k]]>0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=1;
                                    break;
                                }
                            }
                        }
                    }
                }

            }

        }//for i=512
    }//for j=424

}
__global__ void fu_shi(unsigned char * pj, unsigned char * temp_pj, size_t width, size_t height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int j = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int idex = (i) + (j) * width;//0--512*424-1=217087
    int neighbors[8] = {(i-1) + (j-1) * width, (i) + (j-1) * width,(i+1) + (j-1) * width,(i-1) + (j) * width, (i+1) + (j) * width,(i-1) + (j+1) * width,(i) + (j+1) * width,(i+1) + (j+1) * width};

    if ( j < height )
    {
         if ( i < width)
        {
            if (temp_pj[idex]>0)//目标像素
            {
                if (j==0)//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[4]]==0)||(temp_pj[neighbors[6]]==0)||(temp_pj[neighbors[7]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=0;
                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[3]]==0)||(temp_pj[neighbors[5]]==0)||(temp_pj[neighbors[6]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=0;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=3;k<8;k++)
                        {
                            if (temp_pj[neighbors[k]]==0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=0;
                                    break;
                                }
                            }
                        }
                    }
                }
                if (j==(height-1))//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[1]]==0)||(temp_pj[neighbors[2]]==0)||(temp_pj[neighbors[4]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=0;
                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[0]]==0)||(temp_pj[neighbors[1]]==0)||(temp_pj[neighbors[3]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=0;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=0;k<5;k++)
                        {
                            if (temp_pj[neighbors[k]]==0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=0;
                                    break;
                                }
                            }
                        }
                    }
                }

                if ((j>0)&&(j<(height-1)))//上边界
                {
                    if(i==0)
                    {
                        if ((temp_pj[neighbors[1]]==0)||(temp_pj[neighbors[2]]==0)||(temp_pj[neighbors[4]]==0)||(temp_pj[neighbors[6]]==0)||(temp_pj[neighbors[7]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=1;
                        }
                    }
                    if(i==width-1)
                    {
                        if ((temp_pj[neighbors[0]]==0)||(temp_pj[neighbors[1]]==0)||(temp_pj[neighbors[3]]==0)||(temp_pj[neighbors[5]]==0)||(temp_pj[neighbors[6]]==0))//周围有不是黄色像素的
                        {
                            pj[idex]=0;
                        }
                    }
                    if((i>0) && (i<width-1))
                    {
                        int num_=0;
                        for (int k=0;k<8;k++)
                        {
                            if (temp_pj[neighbors[k]]==0)
                            {
                                num_++;
                                if (num_>2)
                                {
                                    pj[idex]=0;
                                    break;
                                }
                            }
                        }
                    }
                }

            }

        }//for i=512
    }//for j=424

            //腐蚀代码


}
__global__ void kernel3(const float* d_dis_depth, const unsigned int* d_raw_rgb, const float* cc, float *_cloud,unsigned char *d_table, unsigned char *seg_result,int *n)//, const float* cc, float *_cloud
{
    int c = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int r = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int index = (c) + (r) * 512;//0--512*424-1=217087
    int depth_to_c_off;
    const float &cx = cc[0];
    const float &cy = cc[1];
    const float &fx = cc[2];
    const float &fy = cc[3];
    if ( r < 424 )
    {
        if ( c < 512)
        {
            const int &id = tex1Dfetch(texref0,index);
            const float z = id <0 ? 0.02f : d_dis_depth[id];
            const int c_off = (z>0)*((tex1Dfetch(texref1,index) + __fdividef(52.0f,z)) * 1081.372070f + 960.0f + tex1Dfetch(texref2,index) * 1920);
            depth_to_c_off = ((z>0)&&(id>=0)&&(c_off>=0)&&(c_off<1920 * 1080)) * (c_off+1) + (-1);
//            const float &depth_v = d_dis_depth[id] * (id>=0);
//            _cloud[index*8+0] = (c-cx) * (fx) * depth_v;
//            _cloud[index*8+1] = (r-cy) * (fy) * depth_v;
//            _cloud[index*8+2] = depth_v;
            const float &depth_v = (id>=0) * d_dis_depth[id];
            _cloud[index*8+0] = ((depth_v>12000.f)||(depth_v<=1.f))? 100000.0f : ((c-cx) * (fx) * depth_v);
            _cloud[index*8+1] = ((depth_v>12000.f)||(depth_v<=1.f))? 100000.0f : ((r-cy) * (fy) * depth_v);
            _cloud[index*8+2] = ((depth_v>12000.f)||(depth_v<=1.f))? 100000.0f : depth_v;
            const int &rgb_val = depth_to_c_off < 0 ? 0 : d_raw_rgb[depth_to_c_off];
            u_char* rgba = (u_char*) ( _cloud + index*8+4 );
            const u_char* bgra = (const u_char*) &rgb_val;
            rgba[0] = bgra[2];
            rgba[1] = bgra[1];
            rgba[2] = bgra[0];
            //颜色分割
            seg_result[index]=d_table[rgba[0]/4*64*64 + rgba[1]/4*64 + rgba[2]/4];
            if(seg_result[index]==0)//黄色像素
            {
                seg_result[index]=1;
                atomicAdd(&(n[0]),1);
            }
            else
                seg_result[index]=0;
        }
    }
    __syncthreads();
}

void registration_kernel(const float* d_dis_depth, const unsigned int* d_raw_rgb, const int* d_map_dist, const float* d_map_x, const int* d_map_yi, const float* cc, float *_cloud, unsigned char *d_table, unsigned char *seg_result, int *num_yellow)//, const float* cc, float *_cloud
{
    hipBindTexture(0,texref0,d_map_dist,imageSize1);
    hipBindTexture(0,texref1,d_map_x,imageSize1);
    hipBindTexture(0,texref2,d_map_yi,imageSize1);

    int *d_n;
    hipMalloc((void**)&d_n,sizeof(int));
    hipMemset(d_n,0,sizeof(int));
    kernel3<<< Grid, threadsPerBlock >>>(d_dis_depth, d_raw_rgb, cc, _cloud, d_table, seg_result, d_n);

    hipUnbindTexture(texref0);
    hipUnbindTexture(texref1);
    hipUnbindTexture(texref2);
    hipMemcpy(num_yellow,d_n,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(d_n);
}
//float badpt = std::numeric_limits<float>::quiet_NaN ();

__global__ void cluster_center_kernel(unsigned char *d_seg_t,unsigned char * segment_table)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int j = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int idex = (i) + (j) * 512;//0--512*424-1=217087
    int neighbors[8] = {(i-1) + (j-1) * 512, (i) + (j-1) * 512,(i+1) + (j-1) * 512,(i-1) + (j) * 512, (i+1) + (j) * 512,(i-1) + (j+1) * 512,(i) + (j+1) * 512,(i+1) + (j+1) * 512};
    if ( j < 424 )
    {
        if ( i < 512)
        {
            //膨胀代码
            if (segment_table[idex]==0)//黄色像素
            {
                for (int k=0; k<8; k++)
                {
                    if ((neighbors[k]>0) && (neighbors[k]<512*424) && (segment_table[neighbors[k]]==1))//周围有不是黄色像素的
                    {
                        d_seg_t[idex]=1;
                        break;
                    }
                }
            }

        }//for i=512
    }//for j=424

}
__global__ void cluster_center_kernel_ex(unsigned char * d_seg_t,unsigned char * segment_table)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//0--511
    int j = threadIdx.y + blockIdx.y * blockDim.y;//0-423
    const int idex = (i) + (j) * 512;//0--512*424-1=217087
    int neighbors[8] = {(i-1) + (j-1) * 512, (i) + (j-1) * 512,(i+1) + (j-1) * 512,(i-1) + (j) * 512, (i+1) + (j) * 512,(i-1) + (j+1) * 512,(i) + (j+1) * 512,(i+1) + (j+1) * 512};
    if ( j < 424 )
    {
        if ( i < 512)
        {
            //腐蚀代码
            if (segment_table[idex]==1)//黄色像素
            {
                for (int k=0; k<8; k++)
                {
                    if ((neighbors[k]>0) && (neighbors[k]<512*424) && (segment_table[neighbors[k]]==0))//周围有不是黄色像素的
                    {
                        d_seg_t[idex]=0;
                        break;
                    }
                }
            }

        }//for i=512
    }//for j=424

}





__global__ void label_initialize(unsigned char * segment_table,  int *d_label, size_t width, size_t height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
     int idex = (i) + (j) * width;//0--512*424-1=217087
    if (idex<width*height)
    {

            if (segment_table[idex]>0  && i>0)
            {
                d_label[idex]=i;
//                atomicAdd(&(ch1[0]),1);
            }
            else
                d_label[idex]=0;
            __syncthreads();
    }
}

__global__ void eight_DLS(unsigned char * segment_table, int *d_label, int *flag)//flag初始值是0，
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 512;//0--512*424-1=217087
    int neighbors[8] = {(i-1) + (j-1) * 512, (i) + (j-1) * 512,(i+1) + (j-1) * 512,(i-1) + (j) * 512, (i+1) + (j) * 512,(i-1) + (j+1) * 512,(i) + (j+1) * 512,(i+1) + (j+1) * 512};
    int no_neigbor=0;
    if (i<512)
    {
        if (j<424)
        {
            if (segment_table[idex]>0)//object pixel
            {
//               atomicAdd(&(ch[0]),1);
                int mini=d_label[idex];
                for (int n=0;n<8;n++)//对8邻域进行搜索
                {
                    while ((neighbors[n]>0)&&(neighbors[n]<512*424)&&(segment_table[neighbors[n]]!=0))//在n方向没有遇到0
                    {
                        //处理某一个方向的pixel
                        if(d_label[neighbors[n]]<mini)
                        {
                            mini=d_label[neighbors[n]];
                        }
                        switch (n) {
                        case 0://左上角
                            neighbors[n]=neighbors[n]-512-1;
                            break;
                        case 1://正上方
                            neighbors[n]=neighbors[n]-512;
                            break;
                        case 2://右上角
                            neighbors[n]=neighbors[n]-512+1;
                            break;
                        case 3://左上角
                            neighbors[n]=neighbors[n]-1;
                            break;
                        case 4://左上角
                            neighbors[n]=neighbors[n]+1;
                            break;
                        case 5://左上角
                            neighbors[n]=neighbors[n]+512-1;
                            break;
                        case 6://左上角
                            neighbors[n]=neighbors[n]+512;
                            break;
                        case 7://左上角
                            neighbors[n]=neighbors[n]+512+1;
                            break;
                        default:
                            break;
                        }
                    }

                }
                if (mini<d_label[idex])
                {
                d_label[idex]=mini;
                flag[0]=1;
                }
            }

//            __syncthreads();
        }
    }

}
__global__ void eight_DLS_last(unsigned char * segment_table,  int *d_label, int *array, int *flag, size_t nnn)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 512;//0--512*424-1=217087
    int s_idex=threadIdx.x +blockDim.x*threadIdx.y;
    int neighbors[8] = {(i-1) + (j-1) * 512, (i) + (j-1) * 512,(i+1) + (j-1) * 512,(i-1) + (j) * 512, (i+1) + (j) * 512,(i-1) + (j+1) * 512,(i) + (j+1) * 512,(i+1) + (j+1) * 512};
//    if (idex==0)
//       printf("aaaaaa %d \n",flag[0]);
    if (i<512)
    {
        if (j<424)
        {
            if ((segment_table[idex]>0) && (d_label[idex]>0))//object pixel
            {
//               if ((nnn>2)&&(d_label[d_label[idex]]==d_label[idex]))
//               {

//               }
//               else
//               {
                int mini=d_label[idex];
                for (int n=0;n<8;n++)//对8邻域进行搜索
                {
                    while ((neighbors[n]>0)&&(neighbors[n]<512*424)&&(segment_table[neighbors[n]]!=0))//在n方向没有遇到0
                    {
                        //处理某一个方向的pixel
                        if((d_label[neighbors[n]]<mini)&&(d_label[neighbors[n]]>0))
                        {
                            mini=d_label[neighbors[n]];
                        }
                        switch (n) {
                        case 0://左上角
                            neighbors[n]=neighbors[n]-512-1;
                            break;
                        case 1://正上方
                            neighbors[n]=neighbors[n]-512;
                            break;
                        case 2://右上角
                            neighbors[n]=neighbors[n]-512+1;
                            break;
                        case 3://左上角
                            neighbors[n]=neighbors[n]-1;
                            break;
                        case 4://左上角
                            neighbors[n]=neighbors[n]+1;
                            break;
                        case 5://左上角
                            neighbors[n]=neighbors[n]+512-1;
                            break;
                        case 6://左上角
                            neighbors[n]=neighbors[n]+512;
                            break;
                        case 7://左上角
                            neighbors[n]=neighbors[n]+512+1;
                            break;
                        default:
                            break;
                        }
                    }

                }
                if (mini<d_label[idex])
                {
                d_label[idex]=mini;
//                atomicExch(&(flag[0]),1);
                }
//               }//if smalllest CCL
            }//if object pixel

                    __syncthreads();
                    __shared__ bool lockx1;
                    __threadfence();
                    if(s_idex==0)
                    {
                        unsigned int lockiii1=atomicAdd(&(array[5]),1);
                        lockx1=(array[5]==224);
                    }
                    __syncthreads();
                    if(lockx1)//保证所有的块均计算完了
                    {

                    }
                    if (nnn==2)
                    {
                    if (d_label[idex]>0)
                    {
                        if ((d_label[idex]!=array[0])&&(d_label[idex]!=array[1])&&(d_label[idex]!=array[2])&&(d_label[idex]!=array[3])&&(d_label[idex]!=array[4]))
                        {
                            for (int k=0;k<5;k++)
                            {
                                if (array[k]==0)
                                {
                                    atomicExch(&(array[k]),d_label[idex]);
                                    break;
                                }
                             }
                         }
                    }

                    }

        }//j=424
    }//i=512

}//函数域
__global__ void eight_DLS_last_obstacle(unsigned char * segment_table,  int *d_label,  size_t width, size_t height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * width;//0--512*424-1=217087
    int s_idex=threadIdx.x +blockDim.x*threadIdx.y;
    int neighbors[8] = {(i-1) + (j-1) * width, (i) + (j-1) * width,(i+1) + (j-1) * width,(i-1) + (j) * width, (i+1) + (j) * width,(i-1) + (j+1) * width,(i) + (j+1) * width,(i+1) + (j+1) * width};
//    if (idex==0)
//       printf("aaaaaa %d \n",flag[0]);
    if (i<width)
    {
        if (j<height)
        {
            if ((segment_table[idex]>0)&&(d_label[idex]>0))//object pixel
            {
//               if ((nnn>2)&&(d_label[d_label[idex]]==d_label[idex]))
//               {

//               }
//               else
//               {
                int mini=d_label[idex];

                for (int n=0;n<8;n++)//对8邻域进行搜索
                {
                    while ((neighbors[n]>0)&&(neighbors[n]<width*height)&&(segment_table[neighbors[n]]!=0))//在n方向没有遇到0
                    {
                        //处理某一个方向的pixel
                        if((d_label[neighbors[n]]<mini)&&(d_label[neighbors[n]]>0))
                        {
                            mini=d_label[neighbors[n]];

                        }
                        switch (n) {
                        case 0://左上角
                            neighbors[n]=neighbors[n]-width-1;
                            break;
                        case 1://正上方
                            neighbors[n]=neighbors[n]-width;
                            break;
                        case 2://右上角
                            neighbors[n]=neighbors[n]-width+1;
                            break;
                        case 3://左上角
                            neighbors[n]=neighbors[n]-1;
                            break;
                        case 4://左上角
                            neighbors[n]=neighbors[n]+1;
                            break;
                        case 5://左上角
                            neighbors[n]=neighbors[n]+width-1;
                            break;
                        case 6://左上角
                            neighbors[n]=neighbors[n]+width;
                            break;
                        case 7://左上角
                            neighbors[n]=neighbors[n]+width+1;
                            break;
                        default:
                            break;
                        }
                    }

                }
                if (mini<d_label[idex])
                {
                d_label[idex]=mini;
//                flag[0]=1;
                }
//               }//if smalllest CCL
            }//if object pixel

        }//j=424
    }//i=512

}//函数域
__global__ void count(int *d_label, int numb, int *center, size_t width, size_t height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * width;//0--512*424-1=217087
    if (idex<width*height)
    {
//        if (idex==0)
//            printf("nnnnnnnnnnnnnnnnnn1111 %d %d\n\n",n[0],numb);
        if (d_label[idex]==numb)
        {
            atomicAdd(&(center[0]),1);
            atomicAdd(&(center[1]),i);
            atomicAdd(&(center[2]),j);
        }
        __syncthreads();
    }
}
double gettime()
{
    double tseconds=0.0;
    struct timeval mytime;
    gettimeofday(&mytime,(struct timezone *)0);
    tseconds=(double)(mytime.tv_sec+mytime.tv_usec*1.0e-6);
    return tseconds;
}
int CCL(unsigned char *d_seg_table, int *d_label, float *ball_position)
{
    double ti=gettime();
    unsigned char *d_seg_table_temp;
    hipMalloc((void **)&d_seg_table_temp,512*424*sizeof(unsigned char));

    for (int j=0;j<3;j++)
    {//corrosion
        hipMemcpy(d_seg_table_temp,d_seg_table,512*424*sizeof(unsigned char),hipMemcpyDeviceToDevice);
        fu_shi<<< Grid, threadsPerBlock >>>(d_seg_table,d_seg_table_temp,512,424);
    }
    for (int i=0;i<4;i++)
    {//dilate
        hipMemcpy(d_seg_table_temp,d_seg_table,512*424*sizeof(unsigned char),hipMemcpyDeviceToDevice);
        peng_zhang<<< Grid, threadsPerBlock >>>(d_seg_table,d_seg_table_temp,512,424);
    }
    hipDeviceSynchronize();
    label_initialize<<< Grid, threadsPerBlock >>>(d_seg_table,d_label,512,424);//
    int *array;
    hipMalloc((void **)&array,6*sizeof(int));
    hipMemset(array,0,6*sizeof(int));
    int *h_array;
    hipHostAlloc((void **)&h_array,5*sizeof(int),hipHostMallocDefault);
    int *d_flag;
    hipMalloc((void **)&d_flag,1*sizeof(int));
    int h_flag=1;

    for (size_t nnn=0;nnn<3;nnn++)
    {
        eight_DLS_last<<< Grid, threadsPerBlock >>>(d_seg_table,d_label,array,d_flag,nnn);
    }
    hipDeviceSynchronize();
    hipMemcpy(h_array,array,5*sizeof(int),hipMemcpyDeviceToHost);//如果change=0,说明没有任何改变，则退出循环
    int *h_center;
    hipHostAlloc((void **)&h_center,3*sizeof(int),hipHostMallocDefault);
    int *center;
    hipMalloc((void **)&center,3*sizeof(int));
    int nu=0;

    for (int h1=0;h1<5;h1++)
    {
        if (h_array[h1]>0)
        {
            hipMemset(center,0,3*sizeof(int));
            count<<< Grid, threadsPerBlock >>>(d_label,h_array[h1],center,512,424);
            hipDeviceSynchronize();
            hipMemcpy(h_center,center,3*sizeof(int),hipMemcpyDeviceToHost);
            if (h_center[0]<20)//如果像素点很少的话忽略这个区域
            {
                h_array[h1]=0;
            }
            else
            {              
                hipMemcpy(h_center,center,2*sizeof(int),hipMemcpyDeviceToHost);
                ball_position[3*nu+0]=h_center[1]/(h_center[0]);
                ball_position[3*nu+1]=h_center[2]/(h_center[0]);
                ball_position[3*nu+2]=h_center[0];
                nu=nu+1;
            }
        }

    }
    hipDeviceSynchronize();
    hipFree(d_flag);
    hipHostFree(h_array);
    hipFree(center);
    hipHostFree(h_center);
    hipFree(array);
    hipFree(d_seg_table_temp);
    return (nu);
}

__global__ void
projection_kernel(float *d_cloud, float *coeffi, int *histo_x, int *histo_y, int *histo_z,unsigned char *d_table)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 512;//0--512*424-1=217087
    __shared__ float coe[12];
    if (threadIdx.x<12)
        coe[threadIdx.x]=coeffi[threadIdx.x];

    if (idex<512*424)
    {
//        if ((d_cloud[idex*8+0]>-6000)&&(d_cloud[idex*8+0]<6000)&&(d_cloud[idex*8+1]>-4000)&&(d_cloud[idex*8+1]<2000)&&(d_cloud[idex*8+2]>500)&&(d_cloud[idex*8+2]<9000))//排除无效点干扰
//        {
            u_char* rgba = (u_char*) ( d_cloud + idex*8+4 );
             //颜色分割
             unsigned char re;
             re=d_table[rgba[0]/4*64*64 + rgba[1]/4*64 + rgba[2]/4];
             if(re!=0)//黄色像素
             {

//                 float xj=d_cloud[idex*8+0];
//                 float yj=d_cloud[idex*8+1];
//                 float zj=d_cloud[idex*8+2];//kinect坐标系
//                 double distance_to_plane = coeffi[0]*xj + (coeffi[1])*yj + (coeffi[2])*zj + coeffi[3]*1000;
                 float xr= coe[0]*d_cloud[idex*8+0]+coe[1]*d_cloud[idex*8+1]+coe[2]*d_cloud[idex*8+2]+coe[3]*1000;//直接转换到机器人坐标系
                 float yr= coe[4]*d_cloud[idex*8+0]+coe[5]*d_cloud[idex*8+1]+coe[6]*d_cloud[idex*8+2]+coe[7]*1000;
                 float zr= coe[8]*d_cloud[idex*8+0]+coe[9]*d_cloud[idex*8+1]+coe[10]*d_cloud[idex*8+2]+coe[11]*1000;
                 //因为相机是向下倾斜的，为了垂直投影到地平面，X方向没差别，Z需要由相机的光轴方向变换到水平方向
                 int x = (int)((yr+10000)/50);
                 int z = (int)((xr)/50);
                 int w=z*400+x;
                 if ((zr>100)&&(zr<300))//&&(distance_to_plane<1)
                 {
                     atomicAdd(&(histo_z[w]),1);
                 }
                 if ((zr>240)&&(zr<860))//&&(distance_to_plane<1)
                 {
                     atomicAdd(&(histo_x[w]),1);
     //                printf("%f %f ",zj_,xr);
                 }
                 if ((zr>1900)&&(zr<2500))//&&(distance_to_plane<1)
                 {
                     atomicAdd(&(histo_y[w]),1);
                 }
             }

//        }

    }
}

//__device__ void kernel_up(int &loc, float *temp_pj)
//{

//}

//__device__ int *location;

__global__ void object_label(unsigned char *pj, int *histo_x, int *histo_y, int *histo_z)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    int histx=histo_x[idex];
    int histy=histo_y[idex];
    int histz=histo_z[idex];
    if (idex<400*240)
    {
//        int z_thresh=
        if (idex<16000)//0-2m
        {
            pj[idex]=(histz>90)&&(histx>10)&&(histy<1)? (histx+histz):0;
        }
        if ((idex>16000)&&(idex<24000))//2-3m
        {
            pj[idex]=(histz>60)&&(histx>2)&&(histy<1)? (histx+histz):0;
        }
        if ((idex<28000)&&(idex>24000))//3.0-3.5m
        {
            pj[idex]=(histz>30)&&(histx>2)&&(histy<1)? (histx+histz):0;
        }
        if ((idex<32000)&&(idex>28000))//3.5-4m
        {
            pj[idex]=(histz>20)&&(histx>1)&&(histy<1)? (histx+histz):0;
        }
        if ((idex<40000)&&(idex>32000))//4-5m
        {
            pj[idex]=(histz>5)&&(histx<50)&&(histy<1)? (histx+histz):0;
        }
        if ((idex<44000)&&(idex>40000))//5-5.5m
        {
            pj[idex]=(histz>3)&&(histx<30)&&(histy<1)? (histx+histz):0;
        }
        if ((idex<56000)&&(idex>44000))//5.5-7m
        {
            pj[idex]=(histz>4)&&(histx<20)&&(histy<1)? (histx+histz):0;
        }
//        if((idex>=56000)&&(idex<60000))
//        {
//          pj[idex]=0;
//        }
//        if ((histo_z[idex]>2)&&(histo_x[idex]>0)&&(histo_y[idex]<2))//之前判断z大于多少的貌似也可以
//        {
//            pj[idex]=histo_x[idex]+histo_z[idex]+histo_y[idex];
//        }
//        else
//        {
//            pj[idex]=0;
//        }
//        pj[idex]=(histo_z[idex]>2)&&(histo_x[idex]>0)&&(histo_y[idex]<10)? (histo_x[idex]+histo_z[idex]+histo_y[idex]) : 0;

    }
}
__global__ void Gaussian1D_kernel(unsigned char *pj, double *kernel, size_t size, unsigned char *result)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    if (idex<400*240)
    {
        int kCenter = floor(size/2.0);
        int nn;
//y  先处理

        for (int n = 0; n < size; n++)
        {
            nn = size - 1 - n;
            int posx = i + (n - kCenter);
            int posy = j;
            int w=posx*400+posy;
            if(posx >= 0 && posx < 400)
            {
                result[idex] += pj[w]*kernel[nn];
            }
        }


    }
}
__global__ void Gaussian1D_kernel_(unsigned char *result, double *kernel, size_t size, unsigned char *pj, int *d_label)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    if (idex<400*240)
    {
        int kCenter = floor(size/2.0);
        int nn;
        if ( (j>9) && (j<230))
        {
                for (int n = 0; n < size; n++)
                {
                    nn = size - 1 - n;
                    int posx = j;
                    int posy = i + (n - kCenter);
                    int w=posy*400+posx;
                    if(posy >= 0 && posy < 240)
                    {
                        pj[idex] += result[w]*kernel[nn];
                    }
                }
//x//后处理

                if ((pj[idex]>0)&&(i>0))
                {
                    d_label[idex]=i;
    //                atomicAdd(&(ch1[0]),1);
                }
        }

    }
}
__global__ void compute_array(int *label,  int *array1)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    if (idex<400*240)
    {
        if (label[idex]>0)
        {

                if ((label[idex]!=array1[0])&&(label[idex]!=array1[1])&&(label[idex]!=array1[2])&&(label[idex]!=array1[3])&&(label[idex]!=array1[4])&&(label[idex]!=array1[5])&&(label[idex]!=array1[6])&&(label[idex]!=array1[7])&&(label[idex]!=array1[8])&&(label[idex]!=array1[9]))
                {
                    for (int k=0;k<10;k++)
                    {
                        if (array1[k]==0)
                        {
                            atomicExch(&(array1[k]),label[idex]);
                            break;
                        }
                     }
               }
        }
    }
}
__global__ void computer_center(int *d_label,int *index,int *t_index,int *amount,int *i_amount,int *j_amount)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    const int sidex =  threadIdx.y;
   __shared__ int region_index[1];
   __shared__ int pixel_amount[1];
   __shared__ int x_amount[1];
   __shared__ int y_amount[1];
   __shared__ unsigned char s_label[192];
   __shared__ int flag[1];
   region_index[0]=0;
   pixel_amount[0]=0;
   x_amount[0]=0;
   y_amount[0]=0;
   flag[0]=0;
   if (sidex<192)
   {
       s_label[sidex]=d_label[idex];
   __syncthreads();
   if (s_label[sidex]>0)
   {
       region_index[0]=s_label[sidex];
       atomicAdd(&(pixel_amount[0]),1);
       atomicAdd(&(x_amount[0]),i);
       atomicAdd(&(y_amount[0]),j);
       flag[0]=1;
   }
   }
   __syncthreads();

   index[blockIdx.x]=flag[0];//0 1 0 1 1 0 0 0 1 1 1为了方便进行求前缀和
   t_index[blockIdx.x]=region_index[0];
   amount[blockIdx.x]=pixel_amount[0];
   i_amount[blockIdx.x]=x_amount[0];
   j_amount[blockIdx.x]=y_amount[0];
   __syncthreads();

}
__global__ void computer_center1(int *output,int *index,int *t_index,int *amount,int *i_amount,int *j_amount,int *output_amount,int *output_x,int *output_y)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;//blockIdx.x=1,blockIdx.y=424,blockDim.x=512,blockDim.y=1
    int j = threadIdx.y + blockIdx.y * blockDim.y;//threadIdx.x=512,threadIdx.y=1
    const int idex = (i) + (j) * 400;//0--512*424-1=217087
    const int sidex =  threadIdx.y;
   int space=1;
   //可能可以用动态并行做
   if(blockIdx.x==0)
   {
       int tt=0;
//       printf("%d ",index[sidex]);//到这儿的时候并不是所有块都完成了前面的index赋值，所以得到的数据不完整
       for (int d=1;d<=400;d=d*2)
       {
           int temp=index[sidex];
           int neighbor=0;
           if ((sidex-space>0))
               neighbor=index[sidex-space];
           __syncthreads();
           if (sidex<space)
               continue;
           else
               tt=temp+neighbor;//index的改变导致neighbor读取的错误
           space=space*2;
           __syncthreads();
           index[sidex]=tt;
       }
   }
   __syncthreads();
//   //index的最后一个数是代表有多少个列有目标值
   if (blockIdx.x==0)
   {

//       int current=sidex;
//       int later=current+1;
       //index这一步没问题。得到的index[399]个数也是对的，下面的判断条件有误
       if (sidex<399)//only use the even(偶数) threads, and the position in index is odd（奇数）.
       {
           if (index[sidex+1]==(index[sidex]+1))
           {
//               printf("even %d %d \n",index[sidex],index[sidex+1]);
               output[index[sidex]]=t_index[sidex+1];
               output_amount[index[sidex]]=amount[sidex+1];
               output_x[index[sidex]]=i_amount[sidex+1];
               output_y[index[sidex]]=j_amount[sidex+1];

           }
       }
   }
   __syncthreads();
   output[99]=index[399];//index最后一个数放的是最终有数值的块的个数
}
int project2D(float *d_cloud,  unsigned char *pj, float *aa,float *obstacle_position,unsigned char *d_table,int *histo_x,int *histo_y,int *histo_z,double *d_kernel,size_t kernel_size,unsigned char *result_, int *label, int *h_label)
{
    int *d_index,*d_t_index,*d_amount,*dx_amount,*dy_amount,*d_output;
    int *output_amount,*output_x, *output_y;
    hipMalloc((void **)&d_index,400*sizeof(int));
    hipMalloc((void **)&d_t_index,400*sizeof(int));
    hipMalloc((void **)&d_amount,400*sizeof(int));
    hipMalloc((void **)&dx_amount,400*sizeof(int));
    hipMalloc((void **)&dy_amount,400*sizeof(int));
    hipMalloc((void **)&d_output,100*sizeof(int));
    hipMalloc((void **)&output_amount,100*sizeof(int));
    hipMalloc((void **)&output_x,100*sizeof(int));
    hipMalloc((void **)&output_y,100*sizeof(int));

    hipMemset(histo_x,0,400*240*sizeof(int));
    hipMemset(histo_y,0,400*240*sizeof(int));
    hipMemset(histo_z,0,400*240*sizeof(int));
    hipMemset(result_,0,400*240*sizeof(unsigned char));
    hipMemset(label,0,400*240*sizeof(int));
    hipMemset(h_label,0,400*240*sizeof(int));

    hipMemset(d_amount,0,400*sizeof(int));
    hipMemset(dx_amount,0,400*sizeof(int));
    hipMemset(dy_amount,0,400*sizeof(int));
    hipMemset(d_index,0,400*sizeof(int));
    hipMemset(d_t_index,0,400*sizeof(int));
    hipMemset(d_output,0,100*sizeof(int));
    hipMemset(output_amount,0,100*sizeof(int));
    hipMemset(output_x,0,100*sizeof(int));
    hipMemset(output_y,0,100*sizeof(int));

    projection_kernel<<< Grid, threadsPerBlock >>>(d_cloud,aa,histo_x,histo_y,histo_z,d_table);
    hipDeviceSynchronize();
    object_label<<< Grid_, threadsPerBlock >>>(pj, histo_x, histo_y, histo_z);
    hipDeviceSynchronize();
    Gaussian1D_kernel<<< Grid_, threadsPerBlock >>>(pj, d_kernel, kernel_size, result_);

    Gaussian1D_kernel_<<< Grid_, threadsPerBlock >>>(result_, d_kernel, kernel_size, pj, label);
    hipDeviceSynchronize();

    for (int nh=0;nh<3;nh++)
    {
        eight_DLS_last_obstacle<<< Grid_, threadsPerBlock >>>(pj,label,400,240);
    }
    hipDeviceSynchronize();
    int *output=new int[100];
    int *amount=new int[100];
    int *x_amount=new int[100];
    int *y_amount=new int[100];

    computer_center<<< Grid1, threadsPerBlock1 >>>(label,d_index,d_t_index,d_amount,dx_amount,dy_amount);
    hipDeviceSynchronize();

    computer_center1<<< Grid1, threadsPerBlock1 >>>(d_output,d_index,d_t_index,d_amount,dx_amount,dy_amount,output_amount,output_x,output_y);
    hipDeviceSynchronize();

    hipMemcpy(output,d_output,100*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(amount,output_amount,100*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(x_amount,output_x,100*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(y_amount,output_y,100*sizeof(int),hipMemcpyDeviceToHost);
    int nu=0;
    vector<int> center_index, center_amount, center_x, center_y;
    if (output[99]>0)
    {
    center_index.push_back(output[0]);
    center_amount.push_back(amount[0]);
    center_x.push_back(x_amount[0]);
    center_y.push_back(y_amount[0]);

    for (int g=1;g<output[99];g++)
    {
        int numb=0;
        for (int l=0;l<center_index.size();l++)
        {
            if (output[g]==center_index.at(l))
            {
                center_amount.at(l)+=amount[g];
                center_x.at(l)+=x_amount[g];
                center_y.at(l)+=y_amount[g];
                break;
            }
            else
            {
                numb++;
                continue;
            }
        }
        if (numb==center_index.size())
        {
            center_index.push_back(output[g]);
            center_amount.push_back(amount[g]);
            center_x.push_back(x_amount[g]);
            center_y.push_back(y_amount[g]);
        }
    }

    for (int hq=0;hq<center_index.size();hq++)
    {
        if (center_amount.at(hq)==0)
            continue;
        float x = center_x.at(hq)/(center_amount.at(hq))*50+(-10000);
        float y = center_y.at(hq)/(center_amount.at(hq))*50;
        float value=-0.02181*y+178.7;
        if (center_amount.at(hq)>4 && center_amount.at(hq)>0.25*value)
        {
           obstacle_position[2*nu+0]=x;
           obstacle_position[2*nu+1]=y;
           nu++;
        }
    }
    }
    hipFree(d_index);
    hipFree(d_t_index);
    hipFree(d_amount);
    hipFree(dx_amount);
    hipFree(dy_amount);
    hipFree(d_output);
    hipFree(output_amount);
    hipFree(output_x);
    hipFree(output_y);

    free(output);
    free(amount);
    free(x_amount);
    free(y_amount);
    return (nu);

}
